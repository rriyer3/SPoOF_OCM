#include "hip/hip_runtime.h"
#include "DLLFunctions_new.cuh"

int main(int argc, char *argv[])
{
	if (argc > 1)
	{
		FILE* ConfigFile = fopen(argv[1], "rt");

		long numX, numY, numx, numy, numF;
		
		long Cent1X, Cent1Y, Cent2X, Cent2Y;
		float Rad1, Rad2;
		uint8_t DFEh = 0;

		char BFileName[512], DataFileName[512], OutputFileName_P1[512], OutputFileName_P2[512], OutputFileName_Raw[512];

		fscanf(ConfigFile, "%s", &DataFileName[0]);
		fscanf(ConfigFile, "%s", &BFileName[0]);
		fscanf(ConfigFile, "%s", &OutputFileName_P1[0]);
		fscanf(ConfigFile, "%s", &OutputFileName_P2[0]);
		fscanf(ConfigFile, "%s", &OutputFileName_Raw[0]);
		fscanf(ConfigFile, "%d", &numX);
		fscanf(ConfigFile, "%d", &numY);
		fscanf(ConfigFile, "%d", &numx);
		fscanf(ConfigFile, "%d", &numy);
		fscanf(ConfigFile, "%d", &numF);
		fscanf(ConfigFile, "%d", &Cent1X);
		fscanf(ConfigFile, "%d", &Cent1Y);
		fscanf(ConfigFile, "%f", &Rad1);
		fscanf(ConfigFile, "%d", &Cent2X);
		fscanf(ConfigFile, "%d", &Cent2Y);
		fscanf(ConfigFile, "%f", &Rad2);

		fclose(ConfigFile);

		uint8_t* Background = (uint8_t*)malloc(sizeof(uint8_t) * numX * numY * 3 / 2);

		if (strcmp(BFileName, "INVALID") != 0)
		{
			FILE* BFID = fopen(BFileName, "rb");
			fread(Background, sizeof(uint8_t), numX * numY * 3 / 2, BFID);
			fclose(BFID);
		}
		else
		{
			long idx = 0;
			for (idx = 0; idx < (numX * numY * 3 / 2); idx++)
			{
				Background[idx] = 0u;
			}
		}
		printf("----------------------------------------------------\n  SPOOF OCM POST PROCESSING LOG FILE  \n----------------------------------------------------\n");

		printf("File name:        %s\n", DataFileName);
		printf("Background name:  %s\n", BFileName);
		printf("Output P1 name:   %s\n", OutputFileName_P1);
		printf("Output P2 name:   %s\n", OutputFileName_P2);
		printf("%d x %d --> %d x %d (%d frames)\n", numX, numY, numx, numy, numF);
		printf("(%d,%d, r = %1.1f) and (%d,%d, r = %1.1f)\n", Cent1X, Cent1Y, Rad1, Cent2X, Cent2Y, Rad2);

		hipError_t Ret1 = FFPSOCM_InitializeEverything(numX, numY, numx, numy, Background, &Rad1, &Rad2);
		printf("Initialized (%s)\n", hipGetErrorString(Ret1));

		uint8_t* Input = (uint8_t*)malloc(sizeof(uint8_t) * numX * numY * 3 / 2);
		hipfftComplex* RawIm = (hipfftComplex*)malloc(sizeof(hipfftComplex) * numX * numY);
		hipfftComplex* Im1 = (hipfftComplex*)malloc(sizeof(hipfftComplex) * numx * numy);
		hipfftComplex* Im2 = (hipfftComplex*)malloc(sizeof(hipfftComplex) * numx * numy);

		long fidx = 0;
		FILE* DFID = fopen(DataFileName, "rb");
		FILE* P1FID = fopen(OutputFileName_P1, "wb");
		FILE* P2FID = fopen(OutputFileName_P2, "wb");
		FILE* RawFID = NULL;
		if (WRITERAWOUT) RawFID = fopen(OutputFileName_Raw, "wb");
		clock_t t = clock(), t2;
		for (fidx = 0; fidx < numF; fidx++)
		{
			fread(Input, sizeof(uint8_t), (numX * numY * 3) / 2, DFID);

			hipError_t Ret2 = FFPSOCM_ProcessFrame(Input, RawIm, Im1, Im2, numX, numY, numx, numy, &Cent1X, &Cent1Y, &Cent2X, &Cent2Y,
				&Rad1, &Rad2, &DFEh);

			fwrite(Im1, sizeof(hipfftComplex), numx * numy, P1FID);
			if (WRITERAWOUT) fwrite(RawIm, sizeof(hipfftComplex), numX * numY, RawFID);
			fwrite(Im2, sizeof(hipfftComplex), numx * numy, P2FID);
			t2 = clock() - t;
			printf("\rFinished %d/%d frames in %1.1f seconds...                        ", fidx, numF, ((float)t2) / CLOCKS_PER_SEC);
		}

		hipError_t Ret3 = FFPSOCM_DestroyEverything();
		printf("\nDestroyed everything (%s)\n", hipGetErrorString(Ret3));
		fclose(DFID);
		fclose(P1FID);
		fclose(P2FID);
		if (WRITERAWOUT) fclose(RawFID);

		free(Input);
		free(RawIm);
		free(Background);
		free(Im1);
		free(Im2);
	}
	return 0;
}